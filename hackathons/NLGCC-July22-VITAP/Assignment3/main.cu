#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <random>
#include <chrono>
#include <sstream>
#include <exception>
#include <string>
#include <cstddef>

#include <hip/hip_runtime.h>

#include "../helpers.hpp"

/************************** CONVOLUTION IMPLEMENTATIONS BEGIN **********************/

namespace cpu_impl {
    template <class T>
    void convolve3d(const T* RESTRICT input, int N, const T* RESTRICT filter, int M, T* RESTRICT output)
    {
        auto K = N - M + 1;
        auto compute_idx = [](auto z, auto y, auto x, int dim) {
            return (z * dim + y) * dim + x;
        };

        for (int start_input_x = 0; start_input_x < N - M + 1; start_input_x++)
        {
            for (int start_input_y = 0; start_input_y < N - M + 1; start_input_y++)
            {
                for (int start_input_z = 0; start_input_z < N - M + 1; start_input_z++)
                {
                    T accumulator = 0;
                    for (int x = 0; x < M; x++)
                    {
                        for (int y = 0; y < M; y++)
                        {
                            for (int z = 0; z < M; z++)
                            {
                                auto w_idx = compute_idx(z, y, x, M);
                                auto i_idx = compute_idx(start_input_z + z, start_input_y + y, start_input_x + x, N);
                                accumulator += filter[w_idx] * input[i_idx];
                            }
                        }
                    }
                    auto out_idx = compute_idx(start_input_z, start_input_y, start_input_x, K);
                    output [out_idx] = accumulator;
                }
            }
        }
    }
}

namespace gpu_impl {

    template <class T> __device__ T mul_ftz(T x, T y) { return x * y; }
    template <> inline __device__ float mul_ftz(float x, float y) {
        float result;
        asm("mul.ftz.f32 %0, %1, %2;" : "=f"(result) : "f"(x), "f"(y));
        return result;
    }

    template <> inline __device__ double mul_ftz(double x, double y) {
        double result;
        asm("mul.ftz.f64 %0, %1, %2;" : "=d"(result) : "d"(x), "d"(y));
        return result;
    }

    template <class T>
    __global__
    void convolve3d(const T* CUDA_RESTRICT input, int N, const T* CUDA_RESTRICT filter, int M, T* CUDA_RESTRICT output)
    {
        auto K = N - M + 1;
        auto compute_idx = [](auto z, auto y, auto x, int dim) {
            return (z * dim + y) * dim + x;
        };

        for (int out_idx = threadIdx.x + blockIdx.x * blockDim.x; out_idx < K * K * K; out_idx += gridDim.x * blockDim.x)
        {
            auto start_input_x = out_idx % K;
            auto start_input_y = (out_idx / K) % K;
            auto start_input_z = out_idx / (K * K);

            T accumulator = 0;
            for (int x = 0; x < M; x++)
            {
                for (int y = 0; y < M; y++)
                {
                    for (int z = 0; z < M; z++)
                    {
                        auto w_idx = compute_idx(z, y, x, M);
                        auto i_idx = compute_idx(start_input_z + z, start_input_y + y, start_input_x + x, N);
                        accumulator += filter[w_idx] * input[i_idx];
                    }
                }
            }
            output[out_idx] = accumulator;
        }
    }
}

/************************** DOT PRODUCT IMPLEMENTATIONS END **********************/

int main ()
{
    using T = double;
    constexpr int N = 128;
    constexpr int M = 9;

    constexpr int output_dim = N - M + 1;

    constexpr T THRESHOLD = 1e-2;

    std::vector<T> input(N * N * N), weights(M * M * M), output_cpu(output_dim * output_dim * output_dim);
    random_fill(std::begin(input), std::end(input));
    random_fill(std::begin(weights), std::end(weights));

    T* d_input, *d_weights, *d_output;
    CHECK_CUDA(hipMalloc(&d_input, input.size() * sizeof(T)));
    CHECK_CUDA(hipMalloc(&d_weights, weights.size() * sizeof(T)));
    CHECK_CUDA(hipMalloc(&d_output, output_cpu.size() * sizeof(T)));

    std::cout << "CPU Evaluation:\n";
    {
        auto cpu_time = benchmark([&] {
            cpu_impl::convolve3d(input.data(), N, weights.data(), M, output_cpu.data());
        });
        std::cout << "\tRunning time: " << to_milliseconds(cpu_time).count() << "ms\n" << std::endl;
    }
    std::cout << std::endl;

    std::vector<T> output_gpu(output_cpu.size());
    std::cout << "GPU Evaluation:\n";
    {
        auto cpu_time = benchmark([&] {
            CHECK_CUDA(hipMemcpy(d_input, input.data(), input.size() * sizeof(T), hipMemcpyHostToDevice));
            CHECK_CUDA(hipMemcpy(d_weights, weights.data(), weights.size() * sizeof(T), hipMemcpyHostToDevice));

            int grid_size = 0, block_size = 0;
            CHECK_CUDA(hipOccupancyMaxPotentialBlockSize(&grid_size, &block_size, gpu_impl::convolve3d<T>, 0));
            gpu_impl::convolve3d<<<grid_size, block_size>>>(d_input, N, d_weights, M, d_output);

            CHECK_CUDA(hipMemcpy(output_gpu.data(), d_output, output_gpu.size() * sizeof(T), hipMemcpyDeviceToHost));
        });
        std::cout << "\tRunning time (incl. memory copy): " << to_milliseconds(cpu_time).count() << "ms\n" << std::endl;
        print_result(std::begin(output_cpu), std::end(output_cpu), std::begin(output_gpu), THRESHOLD);
    }

    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_weights));
    CHECK_CUDA(hipFree(d_output));
    return 0;
}
