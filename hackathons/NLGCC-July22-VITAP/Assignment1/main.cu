#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <random>
#include <chrono>
#include <sstream>
#include <exception>
#include <string>
#include <cstddef>
#include <iomanip>

#include <hip/hip_runtime.h>
#include <thrust/scan.h>

#include "../helpers.hpp"

namespace cpu {
    template <class T>
    void inclusive_scan(const T* RESTRICT input, T* RESTRICT output, int N)
    {
        output[0] = input[0];
        for (int i = 1; i < N; i++)
            output[i] = output[i - 1] + input[i];
    }

    template <class T>
    void inclusive_scan_stl(const T* RESTRICT input, T* RESTRICT output, int N)
    {
        std::inclusive_scan(input, input + N, output);
    }

    template <class T>
    void inclusive_scan_parallel(const T* RESTRICT input, T* RESTRICT output, int N)
    {
        constexpr int ELEMENTS_PER_ITERATION = 1024;

        auto blelloch_scan_power_of_2 = [](T* buffer) {
            // Up-Sweep Step:
            // indices:   0  1  2  3  4  5  6  7
            // input:     1  2  3  4  5  6  7  8        
            // skip-2:       3     7     11    15
            // skip-4:             10          26 
            // skip-8:                         36
            // up-sweeep: 1  3  3  10 5  11 7  36
            //
            // in skip-K, we access [K, K * 2, K * 3, ...] until we exceed N
            // in skip-K, the children of 'i' is 'i' and 'i' - K / 2

            for (int skip = 2; skip <= ELEMENTS_PER_ITERATION; skip *= 2)
                for (int i = skip - 1; i < ELEMENTS_PER_ITERATION; i += skip)
                    buffer[i] += buffer[i - skip / 2];

            auto total = buffer[ELEMENTS_PER_ITERATION - 1];
            buffer[ELEMENTS_PER_ITERATION - 1] = 0;

            // Down-Sweep Step (Exclusive Scan):
            // N = current node, LC(X) = left child of x, RC(X) = right child of X, M(X) = mirror node of X in the up-sweep tree
            //
            // New LC = N
            // New RC = N + LC(M(N))
            //
            // Trick 1: mirror node sits at the same index in the input
            // Trick 2: in skip-N, the right child is at the same index and left child is K indices behind
            // Trick 3: if you go a depth, you have to use skip-(K/2) for children; for example, LC(M(N)) is skip-(K/2) from M(N)
            // 
            // In summary, to recursively update, read N, save LC(M(N)) (K / 2 indices behind), update the LC(N) and then RC(N) (same index as N).
            // We save LC because it shares the index with LC(M(N))) which is required to compute RC(N).
            //
            // indices:    0  1  2  3  4  5  6  7
            // input:      1  3  3  10 5  11 7  36
            // preprocess: 1  3  3  10 5  11 7  0
            // skip-8:              0           10 (N = 0, LC(M(N)) = 10)      
            // skip-4:        0     3     10    21
            // skip-2:     0  1  3  6  10 15 21 28

            for(int skip = ELEMENTS_PER_ITERATION; skip > 1; skip /= 2)
            {
                for (int i = skip - 1; i < ELEMENTS_PER_ITERATION; i += skip)
                {
                    auto current = buffer[i];
                    auto lc_mirror = buffer[i - skip / 2];

                    buffer[i - skip / 2] = current;
                    buffer[i] = current + lc_mirror;
                }
            }

            // convert to inclusive scan
            for (int i = 1; i < ELEMENTS_PER_ITERATION; i++)
                buffer[i - 1] = buffer[i];
            buffer[ELEMENTS_PER_ITERATION - 1] = total;
            return total;
        };
        
        std::vector<T> sub_totals;
        int num_full_iterations = N / ELEMENTS_PER_ITERATION;
        for (int iter = 0; iter < num_full_iterations; iter++)
        {
            auto input_start = input + iter * ELEMENTS_PER_ITERATION;
            auto output_start = output + iter * ELEMENTS_PER_ITERATION;

            for (int i = 0; i < ELEMENTS_PER_ITERATION; i++)
                output_start[i] = input_start[i];
            
            auto sub_total = blelloch_scan_power_of_2(output_start);
            sub_totals.push_back(sub_total);
        }

        if (num_full_iterations * ELEMENTS_PER_ITERATION < N)
        {
            // scan trailing block
            auto starting_offset = num_full_iterations * ELEMENTS_PER_ITERATION;
            auto ending_offset = N;
            auto range_size = ending_offset - starting_offset;

            auto input_start = input + starting_offset;
            auto output_start = output + starting_offset;

            output_start[0] = input_start[0];
            for (int i = 1; i < range_size; i++)
                output_start[i] = output_start[i - 1] + input_start[i];
            sub_totals.push_back(output_start[range_size - 1]);
        }

        if (sub_totals.size() > 1)
        {
            inclusive_scan_parallel(sub_totals.data(), sub_totals.data(), sub_totals.size());
            for (int iter = 1; iter <= sub_totals.size(); iter++)
            {
                auto residual = sub_totals[iter - 1];

                auto starting_offset = iter * ELEMENTS_PER_ITERATION;
                auto ending_offset = min(N, starting_offset + ELEMENTS_PER_ITERATION);
                auto range_size = ending_offset - starting_offset;

                auto input_start = input + starting_offset;
                auto output_start = output + starting_offset;

                for (int i = 0; i < range_size; i++)
                    output_start[i] += residual;
            }
        }            
    }
}

namespace thrust_gpu {
    template <class T>
    void inclusive_scan(const T* CUDA_RESTRICT input, T* CUDA_RESTRICT output, int N)
    {
        thrust::plus<T> plus;
        thrust::inclusive_scan(thrust::device, input, input + N, output, plus);
    }
}

namespace gpu {

    template <class T>
    __global__
    void hillis_steele_scan(const T* CUDA_RESTRICT input, T* CUDA_RESTRICT buffer, T* CUDA_RESTRICT output, int N)
    {
        const T* buf_in = input;
        T* buf_out = buffer;
        for (int scanned_till = 1; scanned_till < N; scanned_till *= 2)
        {
            #pragma unroll 8
            for (int i = threadIdx.x; i < N; i += blockDim.x)
            {
                if (i < scanned_till)
                    buf_out[i] = buf_in[i];
                else
                    buf_out[i] = buf_in[i] + buf_in[i - scanned_till];
            }

            __syncthreads();

            if (buf_out == buffer)
            {
                buf_in = buffer;
                buf_out = output;
            }
            else
            {
                buf_in = output;
                buf_out = buffer;
            }
        }

        if (buf_in != output)
        {
            for (int i = threadIdx.x; i < N; i += blockDim.x)
                output[i] = buffer[i];
        }
    }

    template <class T>
    void inclusive_scan_v1(const T* d_input, T* d_output, int N)
    {
        T* buffer;
        CHECK_CUDA(hipMalloc(&buffer, N * sizeof(T)));
        gpu::hillis_steele_scan<<<1, 1024>>>(d_input, buffer, d_output, N);
        CHECK_CUDA(hipGetLastError());
        CHECK_CUDA(hipFree(buffer));        
    }

    template <class T, int BLOCK_SIZE, int ELEMENTS_PER_ITERATION>
    __global__
    __launch_bounds__(BLOCK_SIZE)
    void blelloch_scan_power_of_2(const T* input, T* sub_totals, T* output, int N)
    {
        auto starting_offset = blockIdx.x * ELEMENTS_PER_ITERATION;
        auto ending_offset = min(N, starting_offset + ELEMENTS_PER_ITERATION);
        auto range_size = ending_offset - starting_offset;
  
        input += starting_offset;
        output += starting_offset;

        constexpr int NUM_BANKS = 32;
        __shared__ T buffer[ELEMENTS_PER_ITERATION + ELEMENTS_PER_ITERATION / NUM_BANKS];
        auto access_buffer = [](int idx) -> T& {
            return buffer[idx + (idx / NUM_BANKS)];
        };

        for (int i = threadIdx.x; i < ELEMENTS_PER_ITERATION; i += BLOCK_SIZE)
            access_buffer(i) = (i < range_size) ? input[i] : 0;        
   
        // Up-Sweep Step:
        // indices:   0  1  2  3  4  5  6  7
        // input:     1  2  3  4  5  6  7  8        
        // skip-2:       3     7     11    15
        // skip-4:             10          26 
        // skip-8:                         36
        // up-sweeep: 1  3  3  10 5  11 7  36
        //
        // in skip-K, we access [K, K * 2, K * 3, ...] until we exceed N
        // in skip-K, the children of 'i' is 'i' and 'i' - K / 2

        #pragma unroll
        for (int skip = 2; skip <= ELEMENTS_PER_ITERATION; skip *= 2)
        {
            __syncthreads();         
            for (int i = threadIdx.x; i < ELEMENTS_PER_ITERATION / skip; i += BLOCK_SIZE)
            {
                auto idx = (i + 1) * skip - 1;
                access_buffer(idx) += access_buffer(idx - skip / 2);
            }
        }

        T total = 0;
        if (threadIdx.x == 0)
        {
            total = access_buffer(ELEMENTS_PER_ITERATION - 1);
            access_buffer(ELEMENTS_PER_ITERATION - 1) = 0;
        }

        // Down-Sweep Step (Exclusive Scan):
        // N = current node, LC(X) = left child of x, RC(X) = right child of X, M(X) = mirror node of X in the up-sweep tree
        //
        // New LC = N
        // New RC = N + LC(M(N))
        //
        // Trick 1: mirror node sits at the same index in the input
        // Trick 2: in skip-N, the right child is at the same index and left child is K indices behind
        // Trick 3: if you go a depth, you have to use skip-(K/2) for children; for example, LC(M(N)) is skip-(K/2) from M(N)
        // 
        // In summary, to recursively update, read N, save LC(M(N)) (K / 2 indices behind), update the LC(N) and then RC(N) (same index as N).
        // We save LC because it shares the index with LC(M(N))) which is required to compute RC(N).
        //
        // indices:    0  1  2  3  4  5  6  7
        // input:      1  3  3  10 5  11 7  36
        // preprocess: 1  3  3  10 5  11 7  0
        // skip-8:              0           10 (N = 0, LC(M(N)) = 10)      
        // skip-4:        0     3     10    21
        // skip-2:     0  1  3  6  10 15 21 28

        #pragma unroll
        for(int skip = ELEMENTS_PER_ITERATION; skip > 1; skip /= 2)
        {
            __syncthreads();

            #pragma unroll
            for (int i = threadIdx.x; i < ELEMENTS_PER_ITERATION / skip; i += BLOCK_SIZE)
            {
                auto idx = (i + 1) * skip - 1;
                auto current = access_buffer(idx);
                auto lc_mirror = access_buffer(idx - skip / 2);

                access_buffer(idx - skip / 2) = current;
                access_buffer(idx)  = current + lc_mirror;
            }
        }

        __syncthreads();

        // convert to inclusive scan
        for (int i = threadIdx.x; i < range_size - 1; i += BLOCK_SIZE)
            output[i] = access_buffer(i + 1);

        if (threadIdx.x == 0)
        {
            output[range_size - 1] = total;
            if (sub_totals)
                sub_totals[blockIdx.x] = total;
        }        
    }

    template <class T, int BLOCK_SIZE, int ELEMENTS_PER_ITERATION>
    __global__
    void correct_block_scans(const T* input, const T* sub_totals, T* output, int N)
    {
        auto starting_offset = (blockIdx.x + 1) * ELEMENTS_PER_ITERATION;
        auto ending_offset = min(N, starting_offset + ELEMENTS_PER_ITERATION);
        auto range_size = ending_offset - starting_offset;

        input += starting_offset;
        output += starting_offset;

        auto residual = sub_totals[blockIdx.x];
        for (int i = threadIdx.x; i < range_size; i += BLOCK_SIZE)
            output[i] += residual;
    }

    template <class T>
    void inclusive_scan_v2(const T* input, T* output, int N)
    {
        constexpr int BLOCK_SIZE = 64;
        constexpr int ELEMENTS_PER_ITERATION = 512;

        int num_iterations = (N + ELEMENTS_PER_ITERATION - 1) / ELEMENTS_PER_ITERATION;
        T* sub_totals = nullptr;
        if (num_iterations > 1)
        {
            CHECK_CUDA(hipMalloc(&sub_totals, num_iterations * sizeof(T)));
        }

        blelloch_scan_power_of_2<T, BLOCK_SIZE, ELEMENTS_PER_ITERATION><<<num_iterations, BLOCK_SIZE>>>(input, sub_totals, output, N);
        CHECK_CUDA(hipGetLastError());

        if (num_iterations > 1)
        {
            inclusive_scan_v2(sub_totals, sub_totals, num_iterations);
            correct_block_scans<T, BLOCK_SIZE, ELEMENTS_PER_ITERATION><<<num_iterations - 1, BLOCK_SIZE>>>(input, sub_totals, output, N);
            CHECK_CUDA(hipGetLastError());
            CHECK_CUDA(hipDeviceSynchronize());
            CHECK_CUDA(hipFree(sub_totals));
        }
    }
}

int main ()
{
    using T = double;
    const int N = 10'000'000;    

    constexpr float THRESHOLD = 0.001;
    
    std::vector<T> input(N);
    random_fill(std::begin(input), std::end(input));

    T* d_input, *d_output;
    CHECK_CUDA(hipMalloc(&d_input, input.size() * sizeof(T)));
    CHECK_CUDA(hipMalloc(&d_output, input.size() * sizeof(T)));

    std::vector<T> output_cpu(N);
    auto cpu_time = benchmark([&input, &output_cpu, N] {
        cpu::inclusive_scan_parallel(input.data(), output_cpu.data(), N);
    });
    std::cout << "[CPU] Running time: " << to_milliseconds(cpu_time).count() << "ms\n";
    std::cout << std::endl;

    std::vector<T> output_thrust(N);
    auto thrust_gpu_time = benchmark([&] {
        CHECK_CUDA(hipMemcpy(d_input, input.data(), input.size() * sizeof(T), hipMemcpyHostToDevice));
        thrust_gpu::inclusive_scan(d_input, d_output, N);
        CHECK_CUDA(hipGetLastError());
        CHECK_CUDA(hipMemcpy(output_thrust.data(), d_output, output_thrust.size() * sizeof(T), hipMemcpyDeviceToHost));
    });
    std::cout << "[thrust] Running time (incl. memory copy): " << to_milliseconds(thrust_gpu_time).count() << "ms\n";
    print_result(std::begin(output_cpu), std::end(output_cpu), std::begin(output_thrust), THRESHOLD);
    std::cout << std::endl;

    // erase previous results
    CHECK_CUDA(hipMemset(d_output, 0, input.size() * sizeof(T)));

    std::vector<T> output_gpu(N);
    auto gpu_time = benchmark([&] {
        CHECK_CUDA(hipMemcpy(d_input, input.data(), input.size() * sizeof(T), hipMemcpyHostToDevice));
        gpu::inclusive_scan_v2(d_input, d_output, N);
        CHECK_CUDA(hipMemcpy(output_gpu.data(), d_output, output_gpu.size() * sizeof(T), hipMemcpyDeviceToHost));
    });
    std::cout << "[GPU] Running time (incl. memory copy): " << to_milliseconds(gpu_time).count() << "ms" << std::endl;
    print_result(std::begin(output_cpu), std::end(output_cpu), std::begin(output_gpu), THRESHOLD);

    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_output));
    return 0;
}