#include <iostream>
#include <vector>
#include <algorithm>
#include <random>
#include <chrono>
#include <sstream>
#include <exception>
#include <string>
#include <cstddef>
#include <iomanip>

#include <hip/hip_runtime.h>
#include <thrust/scan.h>

#include "../helpers.hpp"

namespace cpu {
    template <class T>
    void sort(const T* RESTRICT input, T* RESTRICT output, int N)
    {
        std::copy(input, input + N, output);
        std::sort(output, output + N);
    }

    template <class T>
    void radix_sort(const T* RESTRICT input, T* RESTRICT output, int N)
    {
        std::copy(input, input + N, output);

        constexpr int NUM_ELEMENTS_PER_SEGMENT = 16;
        constexpr int BITS_PER_ITERATION = 4;
        for (int starting_bit = 0; starting_bit < sizeof(T) * 8; starting_bit += BITS_PER_ITERATION)
        {
            bool masks[NUM_ELEMENTS_PER_SEGMENT];
            int pos[NUM_ELEMENTS_PER_SEGMENT + 1];
            T sorted_nums[NUM_ELEMENTS_PER_SEGMENT];

            for (int pattern = (1 << (BITS_PER_ITERATION - 1)) - 1; pattern >= 0; pattern--)
            {
                for (int i = 0; i < NUM_ELEMENTS_PER_SEGMENT; i++)
                {
                    auto value = output[i];
                    auto bits = (value >> starting_bit) & ((1 << (BITS_PER_ITERATION - 1)) - 1);
                    masks[i] = (bits == pattern);
                }

                pos[0] = 0;
                for (int i = 1; i < NUM_ELEMENTS_PER_SEGMENT; i++)
                    pos[i] = pos[i - 1] + masks[i - 1];

                auto total_matches = pos[NUM_ELEMENTS_PER_SEGMENT - 1] + masks[NUM_ELEMENTS_PER_SEGMENT - 1];

                for (int i = 0; i < NUM_ELEMENTS_PER_SEGMENT; i++)
                {
                    if (masks[i])
                        sorted_nums[pos[i]] = output[i];
                }

                // invert masks
                for (int i = 0; i < NUM_ELEMENTS_PER_SEGMENT; i++)
                    masks[i] = !masks[i];
                
                pos[0] = total_matches;
                for (int i = 1; i <= NUM_ELEMENTS_PER_SEGMENT; i++)
                    pos[i] = pos[i - 1] + masks[i - 1];
  
                for (int i = 0; i < NUM_ELEMENTS_PER_SEGMENT; i++)
                {
                    if (masks[i])
                        sorted_nums[pos[i]] = output[i];
                }

                std::cout << "PASS1\n";
                print_range(sorted_nums, sorted_nums + NUM_ELEMENTS_PER_SEGMENT);
                std::cout << '\n';
                std::copy(sorted_nums, sorted_nums + NUM_ELEMENTS_PER_SEGMENT, output);
            }
        }
    }
}

namespace thrust_gpu {
    template <class T>
    void sort(const T* input, T* output, int N)
    {
        thrust::copy(thrust::device, input, input + N, output);
        thrust::sort(thrust::device, output, output + N);
    }
}

namespace gpu {

}

int main ()
{
    using T = int;
    const int N = 16;    

    constexpr float THRESHOLD = 0.001;
    
    std::vector<T> input(N);
    random_fill_integers(std::begin(input), std::end(input));

    T* d_input, *d_output;
    CHECK_CUDA(hipMalloc(&d_input, input.size() * sizeof(T)));
    CHECK_CUDA(hipMalloc(&d_output, input.size() * sizeof(T)));

    std::vector<T> output_cpu(N);
    auto cpu_time = benchmark([&input, &output_cpu, N] {
        cpu::radix_sort(input.data(), output_cpu.data(), N);
    });
    std::cout << "[CPU] Running time: " << to_milliseconds(cpu_time).count() << "ms\n";
    std::cout << std::endl;
print_range(std::begin(output_cpu), std::end(output_cpu));

    std::vector<T> output_thrust(N);
    auto thrust_gpu_time = benchmark([&] {
        CHECK_CUDA(hipMemcpy(d_input, input.data(), input.size() * sizeof(T), hipMemcpyHostToDevice));
        thrust_gpu::sort(d_input, d_output, N);
        CHECK_CUDA(hipGetLastError());
        CHECK_CUDA(hipMemcpy(output_thrust.data(), d_output, output_thrust.size() * sizeof(T), hipMemcpyDeviceToHost));
    });
    std::cout << "[thrust] Running time (incl. memory copy): " << to_milliseconds(thrust_gpu_time).count() << "ms\n";
    print_result(std::begin(output_cpu), std::end(output_cpu), std::begin(output_thrust), THRESHOLD);
    std::cout << std::endl;

    // erase previous results
    CHECK_CUDA(hipMemset(d_output, 0, input.size() * sizeof(T)));

    std::vector<T> output_gpu(N);
    auto gpu_time = benchmark([&] {
        CHECK_CUDA(hipMemcpy(d_input, input.data(), input.size() * sizeof(T), hipMemcpyHostToDevice));
        // gpu::inclusive_scan_v2(d_input, d_output, N);
        CHECK_CUDA(hipMemcpy(output_gpu.data(), d_output, output_gpu.size() * sizeof(T), hipMemcpyDeviceToHost));
    });
    std::cout << "[GPU] Running time (incl. memory copy): " << to_milliseconds(gpu_time).count() << "ms" << std::endl;
    print_result(std::begin(output_cpu), std::end(output_cpu), std::begin(output_gpu), THRESHOLD);

    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_output));
    return 0;
}