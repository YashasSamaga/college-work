#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <random>
#include <chrono>
#include <sstream>
#include <exception>
#include <string>
#include <cstddef>

#include <hip/hip_runtime.h>

#include "../helpers.hpp"

/************************** SPMV IMPLEMENTATIONS BEGIN **********************/

namespace cpu_impl {
    template <class T>
    void spmv_csr(int M, int N, const int* RESTRICT row_ptr, const int* RESTRICT col_idxs, const T* RESTRICT values, const T* RESTRICT x, T* RESTRICT output)
    {
        for (int i = 0; i < M; i++)
        {
            // row i
            auto row_start = row_ptr[i];
            auto row_end = row_ptr[i + 1];

            T accumulator = 0;
            for (int idx = row_start; idx < row_end; idx++)
            {
                auto c = col_idxs[idx];
                auto v = values[idx];
                accumulator += v * x[c];
            }
            output[i] = accumulator;
        }
    }
}

namespace gpu_impl {

    template <class T> __device__ T load_streaming(T* x) { return *x; }
    template <> inline __device__ int load_streaming(int* x) {
        int value;
        asm("ld.global.cs.u32 %0, [%1];" : "=r"(value) : "l"(x));
        return value;
    }

    template <> inline __device__ float load_streaming(float* x) {
        float value;
        asm("ld.global.cs.f32 %0, [%1];" : "=f"(value) : "l"(x));
        return value;
    }

    template <> inline __device__ double load_streaming(double* x) {
        double value;
        asm("ld.global.cs.f64 %0, [%1];" : "=d"(value) : "l"(x));
        return value;
    }
 
    template <class T>
    __global__
    void spmv_csr(int M, int N, const int* CUDA_RESTRICT row_ptr, const int* CUDA_RESTRICT col_idxs, const T* CUDA_RESTRICT values, const T* CUDA_RESTRICT x, T* CUDA_RESTRICT output)
    {
        constexpr int WARP_SIZE = 32;

        int threadId = threadIdx.x + blockIdx.x * blockDim.x;
        int warpId = threadId / WARP_SIZE;
        int laneId = threadId  % WARP_SIZE;
        int num_warps = blockDim.x * gridDim.x / WARP_SIZE;

        #pragma unroll 1
        for (int i = warpId; i < M; i += num_warps)
        {
            // row i
            auto row_start = row_ptr[i];
            auto row_end = row_ptr[i + 1];

            T local_accumulator = 0;

            #pragma unroll 4
            for (int idx = row_start + laneId; idx < row_end; idx += WARP_SIZE)
            {
                auto c = col_idxs[idx];
                auto v = values[idx];
                local_accumulator += v * x[c];
            }

            auto total = warpReduceSum(local_accumulator);
            if (laneId == 0)
                output[i] = total;
        }
    }
}

/************************** SPMV IMPLEMENTATIONS END **********************/

int main ()
{
    using T = double;
    constexpr int M = 4096, N = 4096;

    constexpr T THRESHOLD = 1e-2;
    constexpr float SPARSITY = 0.9;

    // CSR faster for random sparse matrices
    std::vector<int> row_ptr, col_idxs;
    std::vector<T> values;

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(-1, 1);

    row_ptr.push_back(0);
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
            auto prob = dist(gen);
            if (std::abs(prob) > SPARSITY)
            {
                col_idxs.push_back(j);
                values.push_back(dist(gen));
            }
        }
        row_ptr.push_back(values.size());
    }

    std::vector<T> x(N), output_cpu(N);
    random_fill(std::begin(x), std::end(x));

    int* d_row_ptr, *d_col_idxs;
    T *d_values;
    CHECK_CUDA(hipMalloc(&d_row_ptr, row_ptr.size() * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_col_idxs, col_idxs.size() * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_values, values.size() * sizeof(T)));

    T* d_x;
    CHECK_CUDA(hipMalloc(&d_x, x.size() * sizeof(T)));

    T* d_output;
    CHECK_CUDA(hipMalloc(&d_output, output_cpu.size() * sizeof(T)));

    std::cout << "CPU Evaluation:\n";
    {
        auto cpu_time = benchmark([&] {
            cpu_impl::spmv_csr(M, N, row_ptr.data(), col_idxs.data(), values.data(), x.data(), output_cpu.data());
        });
        std::cout << "\tRunning time: " << to_milliseconds(cpu_time).count() << "ms\n";
    }
    std::cout << std::endl;

    std::vector<T> output_gpu(output_cpu.size());
    std::cout << "GPU Evaluation:\n";
    {
        auto cpu_time = benchmark([&] {
            CHECK_CUDA(hipMemcpy(d_row_ptr, row_ptr.data(), row_ptr.size() * sizeof(int), hipMemcpyHostToDevice));
            CHECK_CUDA(hipMemcpy(d_col_idxs, col_idxs.data(), col_idxs.size() * sizeof(int), hipMemcpyHostToDevice));
            CHECK_CUDA(hipMemcpy(d_values, values.data(), values.size() * sizeof(T), hipMemcpyHostToDevice));
            CHECK_CUDA(hipMemcpy(d_x, x.data(), x.size() * sizeof(T), hipMemcpyHostToDevice));

            int grid_size = 0, block_size = 0;
            CHECK_CUDA(hipOccupancyMaxPotentialBlockSize(&grid_size, &block_size, gpu_impl::spmv_csr<T>, 0));
            gpu_impl::spmv_csr<<<grid_size, block_size>>>(M, N, d_row_ptr, d_col_idxs, d_values, d_x, d_output);

            CHECK_CUDA(hipMemcpy(output_gpu.data(), d_output, output_gpu.size() * sizeof(T), hipMemcpyDeviceToHost));
        });
        std::cout << "\tRunning time (incl. memory copy): " << to_milliseconds(cpu_time).count() << "ms\n" << std::endl;
    }
    auto pr = check_result(std::begin(output_cpu), std::end(output_cpu), std::begin(output_gpu), THRESHOLD);
    print_result(std::begin(output_cpu), std::end(output_cpu), std::begin(output_gpu), THRESHOLD);

    CHECK_CUDA(hipFree(d_row_ptr));
    CHECK_CUDA(hipFree(d_col_idxs));
    CHECK_CUDA(hipFree(d_values));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_output));
    return 0;
}