#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <random>
#include <chrono>
#include <sstream>
#include <exception>
#include <string>
#include <cstddef>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <thrust/inner_product.h>
#include <thrust/execution_policy.h>

#include "../helpers.hpp"

/************************** DOT PRODUCT IMPLEMENTATIONS BEGIN **********************/

namespace cpu_impl {
    template <class T>
    void dotproduct(const T* RESTRICT x, const T* RESTRICT y, T* RESTRICT output, int N)
    {
        T sum{0};
        for (int i = 0; i < N; i++)
            sum += x[i] * y[i];
        *output = sum;
    }

    template <class T>
    void dotproduct_stl(const T* RESTRICT x, const T* RESTRICT y, T* RESTRICT output, int N)
    {
        *output = std::inner_product(x, x + N, y, T{0});
    }
}

namespace cublas_impl {
    template <class T>
    void dotproduct(hipblasHandle_t handle, const T* CUDA_RESTRICT x, const T* CUDA_RESTRICT y, T* CUDA_RESTRICT output, int N)
    {
        static_assert(std::is_same<T, float>::value || std::is_same<T, double>::value);
        if constexpr (std::is_same<T, float>::value)
        {
            CHECK_CUBLAS(hipblasSdot(
                handle, N,
                x, 1,
                y, 1,
                output)
            );
        }
        else if constexpr (std::is_same<T, double>::value)
        {
            CHECK_CUBLAS(hipblasDdot(
                handle, N,
                x, 1,
                y, 1,
                output)
            );
        }
    }
}

namespace thrust_impl {
    template <class T>
    void dotproduct(const T* CUDA_RESTRICT x, const T* CUDA_RESTRICT y, T* CUDA_RESTRICT output, int N)
    {
        *output = thrust::inner_product(thrust::device, x, x + N, y, T{0});
    }
}

namespace gpu_impl {
    template <class T>
    __global__
    void fill(T* x, T value, int N)
    {
        for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += blockDim.x * gridDim.x)
            x[i] = value;
    }

    template <class T>
    __global__
    void dotproduct_v1(const T* CUDA_RESTRICT x, const T* CUDA_RESTRICT y, T* CUDA_RESTRICT output, int N)
    {
        for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += blockDim.x * gridDim.x)
            atomicAdd(output, x[i] * y[i]);
    }

    template <class T>
    __global__
    void dotproduct_v2(const T* CUDA_RESTRICT x, const T* CUDA_RESTRICT y, T* CUDA_RESTRICT output, int N)
    {
        T local_sum(0);
        for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += blockDim.x * gridDim.x)
            local_sum += x[i] * y[i];
        atomicAdd(output, local_sum);
    }

    template <class T>
    __global__
    void dotproduct_v3(const T* CUDA_RESTRICT x, const T* CUDA_RESTRICT y, T* CUDA_RESTRICT output, int N)
    {
        T local_sum(0);
        for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += blockDim.x * gridDim.x)
            local_sum += x[i] * y[i];
        auto warpSum = warpReduceSum(local_sum);
        if (threadIdx.x % 32 == 0)
            atomicAdd(output, local_sum);
    }
}

/************************** DOT PRODUCT IMPLEMENTATIONS END **********************/

int main ()
{
    using T = double;
    constexpr int N = 100'000'000;

    constexpr T THRESHOLD = 1e-1;

    std::vector<T> x(N), y(N);
    random_fill(std::begin(x), std::end(x));
    random_fill(std::begin(y), std::end(y));

    T* d_x, *d_y, *d_output;
    CHECK_CUDA(hipMalloc(&d_x, x.size() * sizeof(T)));
    CHECK_CUDA(hipMalloc(&d_y, y.size() * sizeof(T)));
    CHECK_CUDA(hipMalloc(&d_output, sizeof(T)));

    hipblasHandle_t cublas_handle;
    CHECK_CUBLAS(hipblasCreate(&cublas_handle));

    T output_cpu;
    std::cout << "CPU Evaluation:\n";
    {
        auto cpu_time = benchmark([&] {
            cpu_impl::dotproduct(x.data(), y.data(), &output_cpu, N);
        });
        std::cout << "\tRunning time: " << to_milliseconds(cpu_time).count() << "ms\n";
    }
    std::cout << std::endl;

    std::cout << "GPU Memory Transfer Evaluation:\n";
    {
        auto h2d_transfer_time = benchmark([&] {
            CHECK_CUDA(hipMemcpy(d_x, x.data(), x.size() * sizeof(T), hipMemcpyHostToDevice));
            CHECK_CUDA(hipMemcpy(d_y, y.data(), y.size() * sizeof(T), hipMemcpyHostToDevice));
        });
        std::cout << "[H2D] Transfer time: " << to_milliseconds(h2d_transfer_time).count() << "ms" << '\n';

        T dummy;
        auto d2h_transfer_time = benchmark([&] {
            CHECK_CUDA(hipMemcpy(&dummy, d_output, sizeof(T), hipMemcpyDeviceToHost));
        });
        std::cout << "[D2H] Transfer time: " << to_milliseconds(d2h_transfer_time).count() << "ms" << '\n';
    }
    std::cout << std::endl;

    T cublas_output_gpu;
    std::cout << "cuBLAS Evaluation:\n";
    {
        CHECK_CUDA(hipMemcpy(d_x, x.data(), x.size() * sizeof(T), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(d_y, y.data(), y.size() * sizeof(T), hipMemcpyHostToDevice));
        auto cublas_gpu_time = benchmark([&] {
            cublas_impl::dotproduct(cublas_handle, d_x, d_y, d_output, N);
            CHECK_CUDA(hipGetLastError());
            CHECK_CUDA(hipDeviceSynchronize());
        });
        CHECK_CUDA(hipMemcpy(&cublas_output_gpu, d_output, sizeof(T), hipMemcpyDeviceToHost));
        std::cout << "\tKernel Time: " << to_milliseconds(cublas_gpu_time).count() << "ms\n";

        auto error = std::abs(output_cpu - cublas_output_gpu);
        std::cout << "\tOUTPUT >> cpu: " << output_cpu << ", gpu: " << cublas_output_gpu << '\n';
        std::cout << "\tDIFF >> " << error << '\n';
        std::cout << "\tACCURACY TEST: " << error << "; " << (error < THRESHOLD ? "PASSED" : "FAILED") << '\n';
    }
    std::cout << std::endl;

    T output_gpu = 0;
    std::cout << "Custom GPU Implementation Evaluation:\n";
    {
        CHECK_CUDA(hipMemcpy(d_x, x.data(), x.size() * sizeof(T), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(d_y, y.data(), y.size() * sizeof(T), hipMemcpyHostToDevice));
        auto gpu_time = benchmark([&] {
            gpu_impl::fill<<<1, 1>>>(d_output, T(0), 1);
            CHECK_CUDA(hipGetLastError());

            int grid_size = 0, block_size = 0;
            CHECK_CUDA(hipOccupancyMaxPotentialBlockSize(&grid_size, &block_size, gpu_impl::dotproduct_v2<T>, 0));
            gpu_impl::dotproduct_v2<<<grid_size, block_size>>>(d_x, d_y, d_output, N);
            CHECK_CUDA(hipGetLastError());
            
            CHECK_CUDA(hipDeviceSynchronize());
        });
        CHECK_CUDA(hipMemcpy(&output_gpu, d_output, sizeof(T), hipMemcpyDeviceToHost));
        std::cout << "\t[GPU] Kernel Time: " << to_milliseconds(gpu_time).count() << "ms\n";

        auto error = std::abs(output_cpu - output_gpu);
        std::cout << "\tOUTPUT >> cpu: " << output_cpu << ", gpu: " << output_gpu << '\n';
        std::cout << "\tDIFF >> " << error << '\n';
        std::cout << "\tACCURACY TEST: " << error << "; " << (error < THRESHOLD ? "PASSED" : "FAILED") << '\n';
    }

    CHECK_CUBLAS(hipblasDestroy(cublas_handle));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y));
    CHECK_CUDA(hipFree(d_output));
    return 0;
}